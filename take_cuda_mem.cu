#include <chrono>
#include <cmath>
#include <cstdint>
#include <cstdlib>
#include <hip/hip_runtime.h>

#include <exception>
#include <iostream>
#include <memory>
#include <string>
#include <thread>
#include <vector>

#define CUDA_CHECK(cond)                                                       \
  do {                                                                         \
    auto __cond__ = (cond);                                                    \
    if (__cond__ != hipSuccess) {                                             \
      std::string __msg__ = "Runtime error in " #cond;                         \
      __msg__ +=                                                               \
          ", error code is " + std::to_string(static_cast<int>(__cond__));     \
      throw std::runtime_error(__msg__);                                       \
    }                                                                          \
  } while (false)

static __global__ void FillConstantKernel(uint8_t *p, int n) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < n) {
    p[idx] = static_cast<uint8_t>(idx) & 0xFF;
  }
}

static void CudaFree(void *p) { CUDA_CHECK(hipFree(p)); }

static void ThreadMain(int dev_id, uint8_t *p, int n, hipStream_t stream) {
  CUDA_CHECK(hipSetDevice(dev_id));
  size_t idx = 0;
  while (1) {
    int thread_num = 512;
    int grid_num = (n + thread_num - 1) / thread_num;
    int r = rand() % 10 + 1;
    int m = n * r;
    FillConstantKernel<<<grid_num, thread_num, 0, stream>>>(p, m);
    if (idx % 1 == 0) {
      idx = 0;
      CUDA_CHECK(hipStreamSynchronize(stream));

      std::this_thread::sleep_for(std::chrono::microseconds(100 * r));
    }
    ++idx;
  }
}

int main() {
  int dev_cnt = -1;
  CUDA_CHECK(hipGetDeviceCount(&dev_cnt));

  if (dev_cnt <= 0) {
    std::cerr << "Error! Please set CUDA_VISIBLE_DEVICES before running!"
              << std::endl;
    return -1;
  }

  std::cout << "Device number: " << dev_cnt << std::endl;

  // Malloc 4 GiB GPU and launch kernel
  size_t bytes = (static_cast<size_t>(4) << 30);

  std::vector<std::unique_ptr<uint8_t, void (*)(void *)>> ptrs;
  std::vector<hipStream_t> streams;

  ptrs.reserve(dev_cnt);
  streams.resize(dev_cnt);

  for (int i = 0; i < dev_cnt; ++i) {
    CUDA_CHECK(hipSetDevice(i));
    uint8_t *p = nullptr;
    CUDA_CHECK(hipMalloc(&p, bytes));
    ptrs.emplace_back(p, CudaFree);
    CUDA_CHECK(hipStreamCreate(&streams[i]));
  }

  // Launch several kernels and each device
  std::vector<std::thread> threads;
  threads.reserve(dev_cnt);
  for (int i = 0; i < dev_cnt; ++i) {
    auto *p = ptrs[i].get();
    threads.emplace_back(ThreadMain, i, p, static_cast<int>(bytes >> 5),
                         streams[i]);
  }

  for (auto &th : threads) {
    th.join();
  }

  for (auto &stream : streams) {
    CUDA_CHECK(hipStreamSynchronize(stream));
    CUDA_CHECK(hipStreamDestroy(stream));
  }

  return 0;
}
